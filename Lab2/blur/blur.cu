#include "hip/hip_runtime.h"
#include "ppmIO.h"


#define TILE_WIDTH 16

//@@ INSERT CODE HERE
__global__ void blurKernel(float *out, float *in, int width, int height, int blurSize)

{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width-blurSize/2 && x>=blurSize/2 && y < height-blurSize/2 && y >= blurSize/2 )
    {
		int sum = 0;
        int grayOffset = y * width + x;
		for(int i = -blurSize/2; i<=blurSize/2; i++){
			for(int j = -blurSize/2; j<=blurSize/2; j++){
				sum+=in[(y+i) * width + x+j];
			}
		}
		out[grayOffset] = sum/(blurSize*blurSize);

    }else if(x < width && y < height){
		int grayOffset = y * width + x;
		out[grayOffset] = 0;
	}
}
//@@

int main(int argc, char *argv[])
{
	// check if number of input args is correct
	if (argc != 4)
	{
		printf("Wrong number of arguments: exactly 3 arguments needed (input and output .ppm filename with blur size)\n");
		return 0;
	}

	// get blur size
	int blurSize = atoi(argv[3]);

	///////////////////////////////////////////////////////
	//@@ INSERT CODE HERE
	unsigned int width, height;
    getPPMSize(argv[1], &width, &height);
	float *hostInputImageData = (float *)malloc(width * height * sizeof(float));
    readPPM(argv[1], hostInputImageData,true);

	float *deviceInputImageData;
    float *deviceOutputImageData;

    hipMalloc((void **)&deviceInputImageData, width * height  * sizeof(float));
    hipMalloc((void **)&deviceOutputImageData, width * height * sizeof(float));

	hipMemcpy(deviceInputImageData, hostInputImageData, width * height * sizeof(float), hipMemcpyHostToDevice);
	dim3 dimGrid(ceil((float)width / TILE_WIDTH), ceil((float)height / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    blurKernel<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData, width, height, blurSize);


    float *hostOutputImageData = (float *)malloc(width * height * sizeof(float));
    hipMemcpy(hostOutputImageData, deviceOutputImageData, width * height * sizeof(float), hipMemcpyDeviceToHost);
    writePPM(argv[2], hostOutputImageData, width, height, 1);

    free(hostInputImageData);
    free(hostOutputImageData);
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
	///////////////////////////////////////////////////////

	return 0;
}
