/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        printf("Wrong number of arguments: exactly 1 argument needed (number of elements)\n");
        return 1;
    }

    const unsigned int N = atoi(argv[1]);
    const unsigned int bytes = N * sizeof(int);

    int *h_a = (int *)malloc(bytes);
    if (h_a == NULL)
    {
        printf("Cannot allocate host memory!\n");
        return 2;
    }

    int *d_a;
    hipError_t status = hipMalloc((int **)&d_a, bytes);
    if (status != hipSuccess)
    {
        printf("Cannot allocate device memory!\n");
        return 3;
    }

    memset(h_a, 0, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

    return 0;
}